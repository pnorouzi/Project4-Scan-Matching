#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
//#include "utilityCore.hpp"
#include "kernel.h"
#include ""
#include <glm/gtc/type_ptr.hpp>
#include "svd3.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

inline int ilog2(int x) {
	int lg = 0;
	while (x >>= 1) {
		++lg;
	}
	return lg;
}

inline int ilog2ceil(int x) {
	return x == 1 ? 0 : ilog2(x - 1) + 1;
}

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128
#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 0.1f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in ScanMatching::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_color;
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

glm::vec3* dev_first;
glm::vec3* dev_first_buf;
glm::vec3* dev_second;
glm::vec3* dev_corr;
glm::mat3* dev_rot;
glm::vec3* dev_trans;

/*
glm::vec3* dev_first;
glm::vec3* dev_first_buf;
glm::vec3* dev_second;
glm::vec3* dev_corr;
glm::mat3* dev_rot;
glm::vec3* dev_trans;
*/
/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);
	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));

}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernAddColor(int N, glm::vec3* dev_color, glm::vec3 val) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		dev_color[index] = val;
	}
}


__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

void scanmatch::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_color, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}



/**
* Initialize memory, update some globals
*/
void scanmatch::initSimulation(int N_first, int N_second, glm::vec3* first_points, glm::vec3* second_points) {
	int N = N_first + N_second;
	numObjects = N;
	hipMalloc((void**)&dev_pos, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_color, numObjects * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_color failed!");

	hipMalloc((void**)&dev_first, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc failed!");
	hipMemcpy(dev_first, first_points, N_first * sizeof(glm::vec3), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy failed!");

	hipMalloc((void**)&dev_first_buf, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc failed!");
	hipMemcpy(dev_first_buf, first_points, sizeof(glm::vec3) * N_first, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy failed!");

	hipMalloc((void**)&dev_second, N_second * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc failed!");
	hipMemcpy(dev_second, second_points, sizeof(glm::vec3) * N_second, hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy failed!");

	hipMalloc((void**)&dev_corr, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc failed!");

	hipMalloc((void**)&dev_rot, sizeof(glm::mat3));
	checkCUDAErrorWithLine("hipMalloc failed!");

	hipMalloc((void**)&dev_trans, sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc failed!");

	hipMemcpy(dev_pos, first_points, N_first * sizeof(glm::vec3), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy failed!");
	hipMemcpy(dev_pos + N_first, second_points, N_second * sizeof(glm::vec3), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy failed!");

	printf("here");

	dim3 fullBlocksPerGrid1((N_first + blockSize - 1) / blockSize);
	kernAddColor << <fullBlocksPerGrid1, blockSize >> > (N_first, dev_color, glm::vec3(1, 1, 0));

	dim3 fullBlocksPerGrid2((N_second + blockSize - 1) / blockSize);
	kernAddColor << <fullBlocksPerGrid2, blockSize >> > (N_second, dev_color+N_first, glm::vec3(0, 0, 1));

	gridCellWidth = 2.0f;
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;
	hipDeviceSynchronize();
}







__global__ void findmatch(int N_first, int N_second, glm::vec3* dev_first, glm::vec3* dev_second, glm::vec3* dev_corr) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_first) {
		return;
	}

	glm::vec3 desired_point;
	float min_distance = LONG_MAX;
	for (int ind = 0; ind < N_second; ind++) {
		float distance = glm::distance(dev_first[index], dev_second[ind]);
		if (distance < min_distance) {
			desired_point = dev_second[ind];
			min_distance = distance;
		}

	}

	dev_corr[index] = desired_point;
}

__global__ void up_sweep(int N, glm::vec3 *Dev_odata, int d) {

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	index = index * (1 << (d + 1));

	if (index > N - 1) {
		return;
	}

	if (((index + (1 << (d)) - 1) < N) && ((index + (1 << (d + 1)) - 1) < N)) {

		Dev_odata[index + (1 << (d + 1)) - 1].x += Dev_odata[index + (1 << (d)) - 1].x;
		Dev_odata[index + (1 << (d + 1)) - 1].y += Dev_odata[index + (1 << (d)) - 1].y;
		Dev_odata[index + (1 << (d + 1)) - 1].z += Dev_odata[index + (1 << (d)) - 1].z;
	}

}

void find_mean_vec(int n, glm::vec3 *dev_idata, glm::vec3 *dev_mean) {

	//printArray(n, idata);
	//int new_n = n;
	n = 1 << ilog2ceil(n); // make n something that is power of 2

	glm::vec3 *dev_odata;
	hipMalloc((void**)&dev_odata, n * sizeof(glm::vec3));

	hipMemcpy(dev_odata, dev_idata, n * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	for (int d = 0; d <= ((ilog2ceil(n)) - 1); d++) {
		int count_thread = 1 << ((ilog2ceil(n) - d - 1));   // i need ceil(n/d) threads total
		dim3 fullBlocksPerGrid(((count_thread)+blockSize - 1) / blockSize);
		up_sweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
	}
	dev_odata[n - 1] /= n;
	dev_mean = &dev_odata[n - 1];
}

__global__ void Subtract_element(int n, glm::vec3* dev_idata, glm::vec3* mean) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index > n - 1) {
		return;
	}

	dev_idata[index] -= *mean;
}

__global__ void multiply_transpose(int n, glm::vec3* dev_first, glm::vec3* dev_second, glm::mat3 *out) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index > n - 1) {
		return;
	}

	out[index] = glm::outerProduct(dev_first[index], dev_second[index]);
}

__global__ void update(int N_first, glm::vec3 *dev_first, glm::mat3 dev_rot, glm::vec3 dev_trans, glm::vec3* dev_pos) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N_first)
		return;

	dev_pos[index] = dev_rot * dev_first[index] + dev_trans;
}

void scanmatch::run_GPU(int N_first, int N_second) {

	dim3 numBlocks_first((N_first + blockSize - 1) / blockSize);
	dim3 numBlocks_second((N_second + blockSize - 1) / blockSize);
	dim3 numBlocks_rot((3 * 3 + blockSize - 1) / blockSize);
	dim3 numBlocks3_tran((3 * 1 + blockSize - 1) / blockSize);

	findmatch << <numBlocks_first, blockSize >> > (N_first, N_second, dev_first, dev_second, dev_corr);

	glm::vec3 *dev_mean_first;
	glm::vec3 *dev_mean_corr;

	hipMalloc((void**)&dev_mean_first, sizeof(glm::vec3));
	hipMalloc((void**)&dev_mean_corr, sizeof(glm::vec3));

	find_mean_vec(N_first, dev_first, dev_mean_first);
	find_mean_vec(N_first, dev_corr, dev_mean_corr);

	glm::vec3 *dev_centered_first;
	glm::vec3 *dev_centered_corr;

	hipMalloc((void**)&dev_centered_first, N_first * sizeof(glm::vec3));
	hipMalloc((void**)&dev_centered_corr, N_first * sizeof(glm::vec3));

	hipMemcpy(dev_centered_first, dev_first, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_centered_corr, dev_corr, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	Subtract_element << <numBlocks_first, blockSize >> > (N_first, dev_centered_first, dev_mean_first);
	Subtract_element << <numBlocks_first, blockSize >> > (N_first, dev_centered_corr, dev_mean_corr);

	glm::mat3 *dev_B_svds;

	hipMalloc((void**)&dev_B_svds, N_first * sizeof(glm::mat3));

	multiply_transpose << <numBlocks_first, blockSize >> > (N_first, dev_centered_first, dev_centered_corr, dev_B_svds);

	//glm::mat3 *dev_W;

	//hipMalloc((void**)&dev_W, sizeof(glm::mat3));

	glm::mat3 W = thrust::reduce(thrust::device, dev_B_svds, dev_B_svds + N_first, glm::mat3(0));

	//glm::mat3 *dev_W;
	//glm::mat3 U;
	//glm::mat3 S;
	//glm::mat3 V;

	//hipMalloc((void**)&dev_W, sizeof(glm::mat3));

	//hipMemcpy(dev_W, &W, sizeof(glm::mat3), hipMemcpyHostToDevice);

	float U[3][3] = { 0 };
	float S[3][3] = { 0 };
	float V[3][3] = { 0 };

	svd(W[0][0], W[0][1], W[0][2], W[1][0], W[1][1], W[1][2], W[2][0], W[2][1], W[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);


	glm::mat3 host_U(glm::vec3(U[0][0], U[1][0], U[2][0]), glm::vec3(U[0][1], U[1][1], U[2][1]), glm::vec3(U[0][2], U[1][2], U[2][2]));
	glm::mat3 host_Vt(glm::vec3(V[0][0], V[0][1], V[0][2]), glm::vec3(V[1][0], V[1][1], V[1][2]), glm::vec3(V[2][0], V[2][1], V[2][2]));

	glm::vec3 *host_mean_first = new glm::vec3[1];
	glm::vec3 *host_mean_corr = new glm::vec3[1];

	hipMemcpy(host_mean_first, dev_mean_first, sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipMemcpy(host_mean_corr, dev_mean_corr, sizeof(glm::vec3), hipMemcpyDeviceToHost);

	glm::mat3 host_rot = host_U * host_Vt;
	glm::vec3 host_trans = host_mean_corr[0] - host_rot * host_mean_first[0];

	hipMemcpy(dev_rot, &host_rot, sizeof(glm::mat3), hipMemcpyHostToDevice);
	hipMemcpy(dev_trans, &host_trans, sizeof(glm::mat3), hipMemcpyHostToDevice);


	update << <numBlocks_first, blockSize >> > (N_first, dev_first, *dev_rot, *dev_trans, dev_pos);

	hipMemcpy(dev_first, dev_pos, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}











void findmatch_cpu(int N_first, int N_second, glm::vec3* first, glm::vec3* second, glm::vec3* corr) {


	glm::vec3 desired_point;
	float min_distance = LONG_MAX;

	for (int i = 0; i < N_first; i++) {
		for (int j = 0; j < N_second; j++) {
			float distance = glm::distance(first[i], second[j]);
			if (distance < min_distance) {
				desired_point = second[j];
				min_distance = distance;
			}

		}
		corr[i] = desired_point;
	}

}

void find_mean_and_sub(int n, glm::vec3 *idata, glm::vec3 host_mean, glm::vec3 *host_centered) {

	hipMemcpy(host_centered, idata, n * sizeof(glm::vec3), hipMemcpyHostToHost);

	for (int i = 0; i < n; i++) {

		host_mean += idata[i];
	}

	host_mean /= n;



	for (int i = 0; i < n; i++) {
		host_centered[i] -= host_mean;
	}

}

void multiply_transpose_cpu(int n, glm::vec3* first, glm::vec3* corr, glm::mat3 W) {


	float a, b;

	float sum = 0.0f;
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			sum = 0;
			for (int k = 0; k < n; k++) {
				a = (i == 0 ? corr[k].x : i == 1 ? corr[k].y : corr[k].z);
				b = (j == 0 ? first[k].x : j == 1 ? first[k].y : first[k].z);
				sum += a * b;
			}
			W[i][j] = sum;
		}
	}

}

void update_cpu(int N_first, glm::vec3 *host_first, glm::mat3 dev_rot, glm::vec3 dev_trans, glm::vec3* host_first_buf) {


	for (int i = 0; i < N_first; i++) {
		host_first_buf[i] = (dev_rot * host_first[i]) + dev_trans;
	}



}


void scanmatch::run_CPU(int N_first, int N_second, glm::vec3* first_points, glm::vec3* second_points) {

	glm::vec3* host_corr = (glm::vec3*)malloc(N_first * sizeof(glm::vec3*));

	findmatch_cpu(N_first, N_second, first_points, second_points, host_corr);

	glm::vec3 host_mean_first(0.0f, 0.0f, 0.0f);
	glm::vec3 host_mean_corr(0.0f, 0.0f, 0.0f);
	//glm::vec3 *mean = new glm::vec3[1];

	glm::vec3* host_centered_first = (glm::vec3*)malloc(N_first * sizeof(glm::vec3*));

	//*host_centered_first = *first_points;
	glm::vec3* host_centered_corr = (glm::vec3*)malloc(N_first * sizeof(glm::vec3*));
	//*host_centered_corr = host_corr;

	find_mean_and_sub(N_first, first_points, host_mean_first, host_centered_first);
	find_mean_and_sub(N_first, host_corr, host_mean_corr, host_centered_corr);

	glm::mat3 W;

	multiply_transpose_cpu(N_first, host_centered_first, host_centered_corr, W);

	glm::mat3 U;
	//glm::mat3 U = new glm::mat3[1];
	glm::mat3 S;
	glm::mat3 V;

	svd(W[0][0], W[0][1], W[0][2], W[1][0], W[1][1], W[1][2], W[2][0], W[2][1], W[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);

	glm::mat3 host_U(glm::vec3(U[0][0], U[1][0], U[2][0]), glm::vec3(U[0][1], U[1][1], U[2][1]), glm::vec3(U[0][2], U[1][2], U[2][2]));
	glm::mat3 host_Vt(glm::vec3(V[0][0], V[0][1], V[0][2]), glm::vec3(V[1][0], V[1][1], V[1][2]), glm::vec3(V[2][0], V[2][1], V[2][2]));

	glm::mat3 host_rot = host_U * host_Vt;
	glm::vec3 host_trans = host_mean_corr - host_rot * host_mean_first;

	glm::vec3* new_buf = (glm::vec3*)malloc(N_first * sizeof(glm::vec3*));

	hipMemcpy(new_buf, first_points, N_first * sizeof(glm::vec3), hipMemcpyHostToHost);

	update_cpu(N_first, first_points, host_rot, host_trans, new_buf);

	hipMemcpy(first_points, new_buf, N_first * sizeof(glm::vec3), hipMemcpyHostToHost);
	hipMemcpy(dev_pos, new_buf, N_first * sizeof(glm::vec3), hipMemcpyHostToDevice);
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/


/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/


void scanmatch::endSimulation() {
	hipFree(dev_pos);
	hipFree(dev_color);
	hipFree(dev_first);
	hipFree(dev_first_buf);
	hipFree(dev_second);
	hipFree(dev_corr);
	hipFree(dev_rot);
	hipFree(dev_trans);
}
