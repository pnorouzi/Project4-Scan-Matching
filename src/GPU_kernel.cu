#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <hipblas.h>
#include <fstream>
#include <glm/glm.hpp>
#include "svd3.h"
#include "kernel.h"
#include ""
#include "GPU_kernel.h"


#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)


void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#define blockSize 128



glm::vec3* dev_first;
glm::vec3* dev_first_buf;
glm::vec3* dev_second;
glm::vec3* dev_corr;
glm::mat3* dev_rot;
glm::vec3* dev_trans;


void scanmatch::GPU::initSimulation(int N_first, int N_second, glm::vec3 *first, glm::vec3 *second) {

	hipMalloc((void**)&dev_first, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_first failed!");
	hipMemcpy(dev_first, first, sizeof(glm::vec3) * N_first, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_first, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_second failed!");
	hipMemcpy(dev_first_buf, first, sizeof(glm::vec3) * N_first, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_second, N_second * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_second failed!");
	hipMemcpy(dev_second, second, sizeof(glm::vec3) * N_second, hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_corr, N_first * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_corr failed!");

	hipMalloc((void**)&dev_rot, sizeof(glm::mat3));
	checkCUDAErrorWithLine("hipMalloc dev_rot failed!");

	hipMalloc((void**)&dev_trans, sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_trans failed!");

}



__global__ void findmatch(int N_first, int N_second, glm::vec3* dev_first, glm::vec3* dev_second, glm::vec3* dev_corr) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N_first) {
		return;
	}
	
	glm::vec3 desired_point;
	float min_distance = LONG_MAX;
	for (int ind = 0; ind < N_second; ind++) {
		float distance = glm::distance(dev_first[index], dev_second[ind]);
		if (distance < min_distance) {
			desired_point = dev_second[ind];
			min_distance = distance;
		}

	}
	
	dev_corr[index] = desired_point;
}

__global__ void up_sweep(int N, glm::vec3 *Dev_odata, int d) {

	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	index = index * (1 << (d + 1));

	if (index > N - 1) {
		return;
	}

	if (((index + (1 << (d)) - 1) < N) && ((index + (1 << (d + 1)) - 1) < N)) {

		Dev_odata[index + (1 << (d + 1)) - 1].x += Dev_odata[index + (1 << (d)) - 1].x;
		Dev_odata[index + (1 << (d + 1)) - 1].y += Dev_odata[index + (1 << (d)) - 1].y;
		Dev_odata[index + (1 << (d + 1)) - 1].z += Dev_odata[index + (1 << (d)) - 1].z;
	}

}

inline int ilog2(int x) {
	int lg = 0;
	while (x >>= 1) {
		++lg;
	}
	return lg;
}

inline int ilog2ceil(int x) {
	return x == 1 ? 0 : ilog2(x - 1) + 1;
}

void find_mean_vec(int n, glm::vec3 *dev_idata, glm::vec3 *dev_mean) {

	//printArray(n, idata);
	//int new_n = n;
	n = 1 << ilog2ceil(n); // make n something that is power of 2
	
	glm::vec3 *dev_odata;
	hipMalloc((void**)&dev_odata, n * sizeof(glm::vec3));

	hipMemcpy(dev_odata, dev_idata, n * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	for (int d = 0; d <= ((ilog2ceil(n)) - 1); d++) {
		int count_thread = 1 << ((ilog2ceil(n) - d - 1));   // i need ceil(n/d) threads total
		dim3 fullBlocksPerGrid(((count_thread)+blockSize - 1) / blockSize);
		up_sweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
	}
	dev_odata[n - 1] /= n;
	dev_mean = &dev_odata[n - 1];
}

__global__ void Subtract_element(int n,glm::vec3* dev_idata, glm::vec3* mean) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index > n - 1) {
		return;
	}

	dev_idata[index] -= mean;
}

__global__ void multiply_transpose(int n, glm::vec3* dev_first, glm::vec3* dev_second, glm::mat3 *out) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index > n - 1) {
		return;
	}

	out[index] = glm::outerProduct(dev_first[index], dev_second[index]);
}

__global__ void update(int N_first, glm::vec3 *dev_first, glm::mat3 dev_rot, glm::vec3 dev_trans, glm::vec3* dev_first_buf) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= N_first)
		return;

	dev_first_buf[index] = dev_rot * dev_first[index] + dev_trans;
}
/*
// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(hipblasHandle_t &handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda = m, ldb = k, ldc = m;
	 float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
*/
/*
__global__ void matrix_subtraction(float* A, float* B, float* C, int m, int n) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= m*n) {
		return;
	}
	C[index] = A[index] - B[index];
}

__global__ void addTranslation(float* A, float* trans, int num) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num) {
		return;
	}
	A[index * 3 + 0] += trans[0];
	A[index * 3 + 1] += trans[1];
	A[index * 3 + 2] += trans[2];
}

__global__ void setValueOnDevice(float* device_var, int val) {
	*device_var = val;
}


__global__ void find_svd(glm::mat3 &w, glm::mat3 &u, glm::mat3 &s, glm::mat3 &v) {

	svd(w[0][0], w[0][1], w[0][2], w[1][0], w[1][1], w[1][2], w[2][0], w[2][1], w[2][2],
		u[0][0], u[0][1], u[0][2], u[1][0], u[1][1], u[1][2], u[2][0], u[2][1], u[2][2],
		s[0][0], s[0][1], s[0][2], s[1][0], s[1][1], s[1][2], s[2][0], s[2][1], s[2][2],
		v[0][0], v[0][1], v[0][2], v[1][0], v[1][1], v[1][2], v[2][0], v[2][1], v[2][2]);
}
*/
void scanmatch::GPU::run(int N_first, int N_second) {

	dim3 numBlocks_first((N_first + blockSize - 1) / blockSize);
	dim3 numBlocks_second((N_second + blockSize - 1) / blockSize);
	dim3 numBlocks_rot((3 * 3 + blockSize - 1) / blockSize);
	dim3 numBlocks3_tran((3 * 1 + blockSize - 1) / blockSize);
	
	findmatch << <numBlocks_first, blockSize >> > (N_first, N_second, dev_first, dev_second, dev_corr);

	glm::vec3 *dev_mean_first;
	glm::vec3 *dev_mean_corr;

	hipMalloc((void**)&dev_mean_first, sizeof(glm::vec3));
	hipMalloc((void**)&dev_mean_corr, sizeof(glm::vec3));

	find_mean_vec(N_first, dev_first, dev_mean_first);
	find_mean_vec(N_first, dev_corr, dev_mean_corr);
	
	glm::vec3 *dev_centered_first;
	glm::vec3 *dev_centered_corr;

	hipMalloc((void**)&dev_centered_first, N_first* sizeof(glm::vec3));
	hipMalloc((void**)&dev_centered_corr, N_first* sizeof(glm::vec3));

	hipMemcpy(dev_centered_first, dev_first, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_centered_corr, dev_corr, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

	Subtract_element << <numBlocks_first, blockSize >> > (N_first, dev_centered_first, dev_mean_first);
	Subtract_element << <numBlocks_first, blockSize >> > (N_first, dev_centered_corr, dev_mean_corr);

	glm::mat3 *dev_B_svds;

	hipMalloc((void**)&dev_B_svds, N_first * sizeof(glm::mat3));

	multiply_transpose << <numBlocks_first, blockSize >> > (N_first, dev_centered_first, dev_centered_corr, dev_B_svds);

	//glm::mat3 *dev_W;

	//hipMalloc((void**)&dev_W, sizeof(glm::mat3));
	
	glm::mat3 W = thrust::reduce(thrust::device, dev_B_svds, dev_B_svds + N_first, glm::mat3(0));

	//glm::mat3 *dev_W;
	//glm::mat3 U;
	//glm::mat3 S;
	//glm::mat3 V;

	//hipMalloc((void**)&dev_W, sizeof(glm::mat3));

	//hipMemcpy(dev_W, &W, sizeof(glm::mat3), hipMemcpyHostToDevice);

	float U[3][3] = { 0 };
	float S[3][3] = { 0 };
	float V[3][3] = { 0 };
	
	svd(W[0][0], W[0][1], W[0][2], W[1][0], W[1][1], W[1][2], W[2][0], W[2][1], W[2][2],
		U[0][0], U[0][1], U[0][2], U[1][0], U[1][1], U[1][2], U[2][0], U[2][1], U[2][2],
		S[0][0], S[0][1], S[0][2], S[1][0], S[1][1], S[1][2], S[2][0], S[2][1], S[2][2],
		V[0][0], V[0][1], V[0][2], V[1][0], V[1][1], V[1][2], V[2][0], V[2][1], V[2][2]);


	glm::mat3 host_U(glm::vec3(U[0][0], U[1][0], U[2][0]), glm::vec3(U[0][1], U[1][1], U[2][1]), glm::vec3(U[0][2], U[1][2], U[2][2]));
	glm::mat3 host_Vt(glm::vec3(V[0][0], V[0][1], V[0][2]), glm::vec3(V[1][0], V[1][1], V[1][2]), glm::vec3(V[2][0], V[2][1], V[2][2]));

	glm::vec3 *host_mean_first = new glm::vec3[1];
	glm::vec3 *host_mean_corr = new glm::vec3[1];

	hipMemcpy(host_mean_first, dev_mean_first, sizeof(glm::vec3), hipMemcpyDeviceToHost);
	hipMemcpy(host_mean_corr, dev_mean_corr, sizeof(glm::vec3), hipMemcpyDeviceToHost);

	glm::mat3 host_rot = host_U * host_Vt;
	glm::vec3 host_trans = host_mean_corr[0] - host_rot * host_mean_first[0];

	hipMemcpy(dev_rot, &host_rot, sizeof(glm::mat3) , hipMemcpyHostToDevice);
	hipMemcpy(dev_trans, &host_trans, sizeof(glm::mat3), hipMemcpyHostToDevice);


	update << <numBlocks_first, blockSize >> > (N_first, dev_first, *dev_rot, *dev_trans, dev_first_buf);

	hipMemcpy(dev_first, dev_first_buf, N_first * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

